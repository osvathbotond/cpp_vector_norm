#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <stdio.h>
#include <vector>
#include <cmath>
#include <iostream>
#include <fstream>
#include <random>
#include <typeinfo>
#include <thread>
#include <cstdlib>
#include <algorithm>
#include <cassert>
#include <numeric>
#include <string>

#include "norms_cpu.cpp"

static const int NUM_THREADS = 512;

template<typename T>
__global__ void sum_reduction_double(double* vec, double* res, const int n, const bool power, T p) {
    __shared__ double partial_sum[NUM_THREADS];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        if (power) {
            partial_sum[threadIdx.x] = pow(abs(vec[tid]), p);
        }
        else {
            partial_sum[threadIdx.x] = vec[tid];
        }
    }
    else {
        partial_sum[threadIdx.x] = 0.0;
    }

    // Sync the threads to have all of the needed data in the shared memory
    __syncthreads();

    // Do the reduction (example with 8 numbers):
    // a               b       c   d   e f g h
    // a+e             b+f     c+g d+h e f g h
    // a+e+c+g         b+f+c+g c+g d+h e f g h
    // a+e+c+g+b+f+c+g b+f+c+g c+g d+h e f g h
    // And the result is just the 0-th element
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        // We do need to wait for all of the threads to do the sums
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        res[blockIdx.x] = partial_sum[0];
    }

}

template<typename t>
__global__ void sum_reduction_float(float* vec, float* res, const int n, const bool power, t p) {
    __shared__ float partial_sum[NUM_THREADS];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        if (power) {
            partial_sum[threadIdx.x] = pow(abs(vec[tid]), p);
        }
        else {
            partial_sum[threadIdx.x] = vec[tid];
        }
    }
    else {
        partial_sum[threadIdx.x] = 0;
    }

    // Sync the threads to have all of the needed data in the shared memory
    __syncthreads();

    // Do the reduction (example with 8 numbers):
    // a               b       c   d   e f g h
    // a+e             b+f     c+g d+h e f g h
    // a+e+c+g         b+f+c+g c+g d+h e f g h
    // a+e+c+g+b+f+c+g b+f+c+g c+g d+h e f g h
    // And the result is just the 0-th element
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        // We do need to wait for all of the threads to do the sums
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        res[blockIdx.x] = partial_sum[0];
    }

}
// float, because it returns the time and not the result (it's double on the main branch)
template<typename T>
float gpu_lp(double* vec, int vector_length, T p) {
    // Host-side variables
    std::vector <double> pows(vector_length);
    double res;

    size_t bytes = vector_length * sizeof(double);

    // ceil(vector_length / NUM_THREADS)
    int NUM_BLOCKS = (vector_length + NUM_THREADS - 1) / NUM_THREADS;

    // Pointers to the device-side variables
    double* d_vec, * d_res;

    // Cuda event for the device-side timing
    hipEvent_t evt[2];
    for (auto& e : evt) {
        hipEventCreate(&e);
    }

    // Allocate the memory on the GPU and move the vector (with error handling)
    hipError_t err = hipSuccess;
    err = hipMalloc(&d_vec, bytes);
    if (err != hipSuccess) {
        std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipMalloc(&d_res, bytes);;
    if (err != hipSuccess) {
        std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // The first event, to time only the calculations
    hipEventRecord(evt[0]);

    // The first sum-reduction. Each block gives back a number, so the first NUM_BLOCKS elements
    // of the result d_res will have the needed information for us (the partial sums).
    sum_reduction_double << <NUM_BLOCKS, NUM_THREADS >> > (d_vec, d_res, vector_length, true, p);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error in kernel call (during sum reduction): " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // Since a reduction gives us back NUM_BLOCKS elements, we need to do it until NUM_BLOCKS == 1.
    int left = (int)std::ceil(vector_length / (1.0 * NUM_THREADS));
    int NUM_BLOCKS_RED = (int)std::ceil(NUM_BLOCKS / (1.0 * NUM_THREADS));
    while (left > 1) {
        sum_reduction_double << <NUM_BLOCKS_RED, NUM_THREADS >> > (d_res, d_res, left, false, 0);
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cout << "CUDA error in kernel call (during sum reduction): " << hipGetErrorString(err) << "\n";
            return -1;
        }
        left = (int)std::ceil(left / (1.0 * NUM_THREADS));
        NUM_BLOCKS_RED = (int)std::ceil(NUM_BLOCKS_RED / (1.0 * NUM_THREADS));
    }
    // The second event, to time only the calculations
    hipEventRecord(evt[1]);

    // Copying back to the host (only one number; the 0-th element of the d_res), with error handling.
    err = hipMemcpy(&res, d_res, sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // Freeing the memory on the device. Not doing so can cause memory-leak.
    err = hipFree(d_vec);
    if (err != hipSuccess) {
        std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipFree(d_res);
    if (err != hipSuccess) {
        std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    double norm = std::pow(res, 1.0 / p);

    // Wait for the event evt[1]. This is redundant.
    hipEventSynchronize(evt[1]);

    //Calculating the time
    float dt = 0.0f;
    hipEventElapsedTime(&dt, evt[0], evt[1]);

    return dt;
}
// float, because it returns the time and not the result (it's double on the main branch)
template<typename T>
float gpu_lp(float* vec, int vector_length, T p) {
    // Host-side variables
    std::vector <float> pows(vector_length);
    float res;

    size_t bytes = vector_length * sizeof(float);

    // ceil(vector_length / NUM_THREADS)
    int NUM_BLOCKS = (vector_length + NUM_THREADS - 1) / NUM_THREADS;

    // Pointers to the device-side variables
    float* d_vec, * d_res;

    // Cuda event for the device-side timing
    hipEvent_t evt[2];
    for (auto& e : evt) {
        hipEventCreate(&e);
    }

    // Allocate the memory on the GPU and move the vector (with error handling)
    hipError_t err = hipSuccess;
    err = hipMalloc(&d_vec, bytes);
    if (err != hipSuccess) {
        std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipMalloc(&d_res, bytes);;
    if (err != hipSuccess) {
        std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // The first event, to time only the calculations
    hipEventRecord(evt[0]);

    // The first sum-reduction. Each block gives back a number, so the first NUM_BLOCKS elements
    // of the result d_res will have the needed information for us (the partial sums).
    sum_reduction_float << <NUM_BLOCKS, NUM_THREADS >> > (d_vec, d_res, vector_length, true, p);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error in kernel call (during sum reduction): " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // Since a reduction gives us back NUM_BLOCKS elements, we need to do it until NUM_BLOCKS == 1.
    int left = (int)std::ceil(vector_length / (1.0 * NUM_THREADS));
    int NUM_BLOCKS_RED = (int)std::ceil(NUM_BLOCKS / (1.0 * NUM_THREADS));
    while (left > 1) {
        sum_reduction_float<<<NUM_BLOCKS_RED, NUM_THREADS>>>(d_res, d_res, left, false, 0);
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cout << "CUDA error in kernel call (during sum reduction): " << hipGetErrorString(err) << "\n";
            return -1;
        }
        left = (int)std::ceil(left / (1.0 * NUM_THREADS));
        NUM_BLOCKS_RED = (int)std::ceil(NUM_BLOCKS_RED / (1.0 * NUM_THREADS));
    }
    // The second event, to time only the calculations
    hipEventRecord(evt[1]);

    // Copying back to the host (only one number; the 0-th element of the d_res), with error handling.
    err = hipMemcpy(&res, d_res, sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    // Freeing the memory on the device. Not doing so can cause memory-leak.
    err = hipFree(d_vec);
    if (err != hipSuccess) {
        std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipFree(d_res);
    if (err != hipSuccess) {
        std::cout << "Error freeing allocation: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    double norm = std::pow(res, 1.0 / p);

    // Wait for the event evt[1]. This is redundant.
    hipEventSynchronize(evt[1]);

    //Calculating the time
    float dt = 0.0f;
    hipEventElapsedTime(&dt, evt[0], evt[1]);

    return dt;
}

template<typename T>
void generate_double_times(int cpu_threads, int total_runs, int vector_length, T p) {
    // Writing to a text file
    std::ofstream output;

    std::string fn = std::string("double_p") + std::to_string(p) + std::string("_n") +
        std::to_string(vector_length) + std::string(".dat");
    output.open(fn);
    output << "# The first column contains only the calculation times (power and sum done on the cpu) with " << cpu_threads << "threads, the second "
           << "column contains the total cpu time with the same threads, the third column contains the total gpu times (with copy, malloc, etc) and the "
           <<"fourth column contains only the calculation times (power and sum done on the gpu) on the gpu. \n";
    for (int run = 0; run < total_runs; run++) {
        std::vector <double> vec(vector_length);

        // Not the most effective way of creating a random vector.
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<> dist(-1.0, 1.0);
        for (int i = 0; i < vector_length; i++) {
            vec[i] = dist(e2);
        }

        // Calculating the total times, on the host-side
        auto start_cpu = std::chrono::steady_clock::now();
        auto elapsed_seconds_cpu_calc = parallel_lp(vec, vector_length, p, cpu_threads);
        auto finish_cpu = std::chrono::steady_clock::now();
        double elapsed_seconds_cpu_total = std::chrono::duration_cast<std::chrono::duration<double>>(finish_cpu - start_cpu).count();

        auto start_gpu = std::chrono::steady_clock::now();
        auto res_gpu = gpu_lp(vec.data(), vector_length, p);
        auto finish_gpu = std::chrono::steady_clock::now();
        double elapsed_seconds_gpu_calc = res_gpu / 1000.0;
        double elapsed_seconds_gpu_total = std::chrono::duration_cast<std::chrono::duration<double>>(finish_gpu - start_gpu).count();


        output << elapsed_seconds_cpu_calc << ", " << elapsed_seconds_cpu_total << ", " << elapsed_seconds_gpu_total << ", " << elapsed_seconds_gpu_calc << "\n";
    }
    output.close();
}

template<typename T>
void generate_float_times(int cpu_threads, int total_runs, int vector_length, T p) {
    // Writing to a text file
    std::ofstream output;

    std::string fn = std::string("float_p") + std::to_string(p) + std::string("_n") +
        std::to_string(vector_length) + std::string(".dat");
    output.open(fn);
    output << "# The first column contains only the calculation times (power and sum done on the cpu) with " << cpu_threads << "threads, the second "
           << "column contains the total cpu time with the same threads, the third column contains the total gpu times (with copy, malloc, etc) and the "
           << "fourth column contains only the calculation times (power and sum done on the gpu) on the gpu. \n";
    for (int run = 0; run < total_runs; run++) {
        std::vector <float> vec(vector_length);

        // Not the most effective way of creating a random vector.
        std::random_device rd;
        std::mt19937 e2(rd());
        std::uniform_real_distribution<> dist(-1.0f, 1.0f);
        for (int i = 0; i < vector_length; i++) {
            vec[i] = float(dist(e2));
        }

        // Calculating the total times, on the host-side
        auto start_cpu = std::chrono::steady_clock::now();
        auto elapsed_seconds_cpu_calc = parallel_lp(vec, vector_length, p, cpu_threads);
        auto finish_cpu = std::chrono::steady_clock::now();
        double elapsed_seconds_cpu_total = std::chrono::duration_cast<std::chrono::duration<double>>(finish_cpu - start_cpu).count();

        auto start_gpu = std::chrono::steady_clock::now();
        auto res_gpu = gpu_lp(vec.data(), vector_length, p);
        auto finish_gpu = std::chrono::steady_clock::now();
        double elapsed_seconds_gpu_calc = res_gpu / 1000.0;
        double elapsed_seconds_gpu_total = std::chrono::duration_cast<std::chrono::duration<double>>(finish_gpu - start_gpu).count();


        output << elapsed_seconds_cpu_calc <<", " << elapsed_seconds_cpu_total << ", " << elapsed_seconds_gpu_total << ", " << elapsed_seconds_gpu_calc << "\n";
    }
    output.close();
}

int main() {
    const int cpu_threads = 12;
    const int total_runs = 100;
    const int vector_length = 10'000'000;

    for (int i = 1; i <= 5; i++) {
        generate_double_times(cpu_threads, total_runs, vector_length, i);
        std::cout << "double " << i << " is done. \n";

        generate_float_times(cpu_threads, total_runs, vector_length, i);
        std::cout << "float " << i << " is done. \n";
    }

}